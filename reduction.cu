

#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <limits.h>
#include <stdlib.h>

void run_test();
void printArr(int *arr);

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__inline__ __device__
float warpReduceSum(float val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2) 
    val += __shfl_down(val, offset);
  return val;
}

__inline__ __device__
float blockReduceSum(float val) {

  static __shared__ float shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSum(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

  if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

  return val;
}

__global__ void reduction(float *in, float *out, int N, int s1, int s2, int splane, int dim1, int dim2, int planeCount, int noEls)
{
	float sum =0;
	int cur_plane;

	int area = dim1 * dim2;
	int start = blockIdx.x * noEls * blockDim.x + threadIdx.x;
	int gridStride = blockDim.x * noEls * gridDim.x;

	//relative index and coordinates calculation
	
	
	
	
	/*int tempDiv = area;
	/*
	for(int dimIter=0; dimIter<noDims; dimIter++)
	{
		if(dimIter != noDims-1)
		{
			dCoord = start / tempDiv % dimSizes[dimIter];
		}
	}
	*/
	int target = 0;
	int counter = 0;
	int quarter = dim2 / noEls;
	quarter = quarter *s2;
	for(int i = start;
		i < N;
		i += gridStride)
	{
		sum = 0;
		
		//float sum =0;
		//calculate the first target index
 		

 		
		target = (i%dim1) * s1;
 		target += ((i/ dim1) % dim2) * s2;
 		target += ((i/area)) * splane;
 		//printf("Test: tid= %d  target= %d target2= %d \n\n", i, target, target + (dim2/2 * s2));
 		
 		for(int iter=0; iter < noEls; iter++)
 		{
 			sum += in[target + iter*quarter];

 		}
 		//__syncthreads();
 		/*
 		sum = in[gridStride*counter + target] + in[gridStride*counter + target + quarter] +
 				in[gridStride*counter + target + 2*quarter] + in[gridStride*counter+target+ 3*quarter];
 		*/		
 		//sum += in[i] + in[i+blockDim.x];
 		sum = blockReduceSum(sum);
		if(threadIdx.x == 0)
			out[counter*gridDim.x + blockIdx.x] = sum;
 		
 		counter++;
 		//sum += in[i] + in[i + blockDim.x];
	}
	
}



int main(void)
{
	{
		run_test();
	}
	return 0;
}

void printArr(int *arr)
{
	int i;
	printf("Stride values in order: ");
	for(i=0;i<=sizeof(arr)/sizeof(int);i++)
	{

		printf("%d ", arr[i]);
	}
	printf("\n\n");
}
void run_test()
{

	
	const int dim_len = 3;

	//dimension sizes
	int dims[dim_len] = {128,4096,128};
	//dimensions to reduce
	int rdims[2] = {0,2}; //x and y


	int strides[dim_len];

	strides[0] = 1;

	//total number of elements
	int N = dims[0];




	for(int i=1; i<dim_len; i++){
		strides[i] = dims[i-1] * strides[i-1]; 
		//update N
		N *= dims[i];
	}
	printf("Number of elements: %d\n\n", N);
	printArr(strides);

	//Allocate memory for in and out in host and fill in
	float *in, *out, *d_in, *d_out;
	//int *d_strides, *d_rdims, *d_dims;

	in = (float*)malloc(N*sizeof(float));

	
	int planeCount = 4096;//8192;//131072;

	out = (float*)malloc(planeCount*sizeof(float)); 
	srand(time(NULL));
	for(int i=0; i<N;i++)
	{
		if(i%4096 == 1)
		{
			in[i] = float(i)/1000; //(float)rand() / (float)RAND_MAX;//
		}
		else
		{
			in[i] = float(i)/1000;
		}
	}
	/*
	for(int i=0;i<N;i++)
	{
		printf("%.1f ", in[i]);
	}
	*/
	printf("\n\n");
	//Allocate memory for in and out on device
	hipMalloc(&d_in, N*sizeof(float));
	hipMalloc(&d_out, planeCount*sizeof(float));
	

	//Event variables
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	
	//Transfer host data to device

	gpuErrchk(hipMemcpy(d_in, in, N*sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_out, out, planeCount*sizeof(float), hipMemcpyHostToDevice));


	int s1 = strides[rdims[0]];
	int s2 = strides[rdims[1]];
	int splane = strides[1];
	int dim1 = dims[rdims[0]];
	int dim2 = dims[rdims[1]];
	int noEls = 16;
	//Record kernel
	int noMeasures = 10; //number of measurements to take
	hipEventRecord(start);
	for(int mesIter=0; mesIter<noMeasures;mesIter++)
	{
		reduction<<<2048,1024>>>(d_in,d_out, N, s1, s2, splane, dim1, dim2, planeCount, noEls);
	}
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms,start,stop);

	gpuErrchk(hipMemcpy(out, d_out, planeCount*sizeof(float), hipMemcpyDeviceToHost));
	ms = ms/noMeasures;
	double total = (N+planeCount)*4;

	double ebw = total/(ms*1e6);

	printf("EBW: %f\n", ebw);
	//Check errors
	printf("Strides: %d %d ", s1,s2);
	printf("Plane Stride: %d\n ", splane);
	printf("Plane Count: %d\n", planeCount);

	for(int i=0;i<4;i++)
	{
		
		printf("%.3f %d ", out[i], i);
	}
	printf("%.3f", out[63/*131071*/]);
	printf("\n");
	hipFree(d_in);
	hipFree(d_out);
}
