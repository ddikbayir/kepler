

#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <limits.h>
#include <stdlib.h>

void run_test();
void printArr(int *arr);

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__inline__ __device__
float warpReduceSum(double val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2) 
    val += __shfl_down(val, offset);
  return val;
}

__inline__ __device__
float blockReduceSum(double val) {

  static __shared__ double shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSum(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

  if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

  return val;
}

__global__ void reduction(double *in, double *out, int N, int s1, int s2, int splane, int dim1, int dim2, int planeCount, int noEls)
{
	double sum =0;
	int cur_plane;
	
	int area = dim1 * dim2;
	int start = blockIdx.x * noEls * blockDim.x + threadIdx.x;
	//int gridStride = splane * gridDim.x;
	int gridStride = splane * gridDim.x;
	
	//relative index and coordinates calculation
	int target = 0;
	
	target = (start%dim1) * s1;
 	target += ((start/ dim1) % dim2) * s2;
 	target += ((start/area)) * splane;
	
	
	/*int tempDiv = area;
	/*
	for(int dimIter=0; dimIter<noDims; dimIter++)
	{
		if(dimIter != noDims-1)
		{
			dCoord = start / tempDiv % dimSizes[dimIter];
		}
	}
	*/
	
	int counter = 0;
	int quarter = dim2 / noEls;
	quarter = quarter *s2;
	for(int i = target;
		counter < planeCount/gridDim.x;
		i += gridStride)
	{
		sum = 0;
		
		//float sum =0;
		//calculate the first target index
 		

 		/*
		target = (i%dim1) * s1;
 		target += ((i/ dim1) % dim2) * s2;
 		target += ((i/area)) * splane;
 		*/
 		//printf("Test: tid= %d  target= %d target2= %d \n\n", i, target, target + (dim2/2 * s2));
 		
 		for(int iter=0; iter < noEls; iter++)
 		{
 			sum += in[i + iter*quarter];

 		}
 		//__syncthreads();
 		/*
 		sum = in[gridStride*counter + target] + in[gridStride*counter + target + quarter] +
 				in[gridStride*counter + target + 2*quarter] + in[gridStride*counter+target+ 3*quarter];
 		*/		
 		//sum += in[i] + in[i+blockDim.x];
 		sum = blockReduceSum(sum);
		if(threadIdx.x == 0)
			out[counter*gridDim.x + blockIdx.x] = sum;
 		
 		counter++;
 		//sum += in[i] + in[i + blockDim.x];
	}
	
}

void run_test(int noEls, int noOfBlocks, int r1, int r2, int rplane, int dimen1, int dimen2, int dimen3, int dimen4)
{

	
	const int dim_len = 4;

	//dimension sizes
	int dims[dim_len] = {dimen1,dimen2,dimen3, dimen4};
	//dimensions to reduce
	int rdims[2] = {r1,r2}; //x and y


	int strides[dim_len];

	strides[0] = 1;

	//total number of elements
	int N = dims[0];




	for(int i=1; i<dim_len; i++){
		strides[i] = dims[i-1] * strides[i-1]; 
		//update N
		N *= dims[i];
	}
	printf("Number of elements: %d\n\n", N);
	printArr(strides);

	//Allocate memory for in and out in host and fill in
	double *in, *out, *d_in, *d_out;
	//int *d_strides, *d_rdims, *d_dims;

	in = (double*)malloc(N*sizeof(double));

	
	int planeCount = N/(dims[rdims[0]]*dims[rdims[1]]);//dims[rplane] * dims[3];//8192;//131072;
	/*
	printf("Dimz: %d\n\n", dims[rplane]);
	printf("Dimt: %d\n\n", dims[3]);

	printf("PlaneCount: %d\n\n", planeCount);
	*/
	out = (double*)malloc(planeCount*sizeof(double)); 
	srand(time(NULL));
	for(int i=0; i<N;i++)
	{
		if((i/128)%2048 == 2047)
		{
			in[i] = double(i)/1000; //(float)rand() / (float)RAND_MAX;//
		}
		else
		{
			in[i] = double(i)/1000;
		}
	}
	/*
	for(int i=0;i<N;i++)
	{
		printf("%.1f ", in[i]);
	}
	*/
	printf("\n\n");
	//Allocate memory for in and out on device
	hipMalloc(&d_in, N*sizeof(double));
	hipMalloc(&d_out, planeCount*sizeof(double));
	

	//Event variables
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	
	//Transfer host data to device

	gpuErrchk(hipMemcpy(d_in, in, N*sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_out, out, planeCount*sizeof(double), hipMemcpyHostToDevice));


	int s1 = strides[rdims[0]];
	int s2 = strides[rdims[1]];
	int splane = strides[rplane];
	int dim1 = dims[rdims[0]];
	int dim2 = dims[rdims[1]];
	//int noElems = noEls;
	//Record kernel
	int noMeasures = 10; //number of measurements to take
	hipEventRecord(start);
	for(int mesIter=0; mesIter<noMeasures;mesIter++)
	{
		reduction<<<noOfBlocks,((dim1*dim2)/noEls)>>>(d_in,d_out, N, s1, s2, splane, dim1, dim2, planeCount, noEls);
	}
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms,start,stop);

	gpuErrchk(hipMemcpy(out, d_out, planeCount*sizeof(double), hipMemcpyDeviceToHost));
	ms = ms/noMeasures;
	double total = (N)*8;

	double ebw = total/(ms*1e6);

	printf("EBW: %f\n", ebw);
	//Check errors
	printf("Strides: %d %d ", s1,s2);
	printf("Plane Stride: %d\n ", splane);
	printf("Plane Count: %d\n", planeCount);

	for(int i=0;i<4;i++)
	{
		
		printf("%.3f %d ", out[i], i);
	}
	double sizeOut = sizeof(out);
	double sizeD = sizeof(double);
	int lengthOut = sizeOut/sizeD;
	printf("Length: %d\n", lengthOut);
	printf("%.3f", out[2047/*131071*/]);
	for(int i=0;i<16;i++)
	{
		if(out[i] == 0)
		{
			printf("Incorrect : %d\n", i );
		}
	}
	printf("\n");
	hipFree(d_in);
	hipFree(d_out);
}

int main(int argc, char *argv[])
{
	{
		int noEls = 8;
		int noOfBlocks = 512;
		int r1 = 0;
		int r2 = 1;
		int rplane = 2;
		int dim1 = 32;
		int dim2 = 32;
		int dim3 = 16;
		int dim4 = 4096;

		if(argc > 1)
		{
			noEls = atoi(argv[1]);
			noOfBlocks = atoi(argv[2]);
			r1 = atoi(argv[3]);
			r2 = atoi(argv[4]);
			rplane = atoi(argv[5]);
			dim1 = atoi(argv[6]);
			dim2 = atoi(argv[7]);
			dim3 = atoi(argv[8]);
			dim4 = atoi(argv[9]);
		}
		run_test(noEls, noOfBlocks, r1, r2, rplane, dim1, dim2, dim3, dim4);
	}
	return 0;
}

void printArr(int *arr)
{
	int i;
	printf("Stride values in order: ");
	for(i=0;i<=sizeof(arr)/sizeof(int);i++)
	{

		printf("%d ", arr[i]);
	}
	printf("\n\n");
}

