

#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <limits.h>
#include <stdlib.h>

void run_test();
void printArr(int *arr);

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__inline__ __device__
float warpReduceSum(double val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2) 
    val += __shfl_down(val, offset);
  return val;
}

__inline__ __device__
float blockReduceSum(double val) {

  static __shared__ double shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSum(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

  if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

  return val;
}

__global__ void reduction(double *in, double *out, int N, int s1, int s2, int s3, int splane, int dim1, int dim2, int dim3, int planeCount, int noEls)
{
	double sum =0;
	int cur_plane;
	
	int area = dim1*dim2*dim3;
	int start = blockIdx.x * noEls * blockDim.x + threadIdx.x;
	//int gridStride = splane * gridDim.x;
	int gridStride = splane * gridDim.x;
	
	//relative index and coordinates calculation
	int target = 0;
	
	target = (start%dim1) * s1;
 	target += ((start/ dim1) % dim2) * s2;
 	target += ((start/dim1*dim2) % dim3) * s3;
 	target += ((start/area)) * splane;
	
	
	/*int tempDiv = area;
	/*
	for(int dimIter=0; dimIter<noDims; dimIter++)
	{
		if(dimIter != noDims-1)
		{
			dCoord = start / tempDiv % dimSizes[dimIter];
		}
	}
	*/
	
	int counter = 0;
	int quarter = (blockDim.x%dim1) * s1;
	quarter += ((blockDim.x/dim1) %dim2) * s2;
	quarter = (((blockDim.x/(dim1*dim2))) %dim3) * s3;
	//quarter = quarter * s3;
	//int quarter = blockDim.x * s1;
	//quarter = quarter *s2;
	//int step = s1; //step of the first reduction dimension
	for(int i = target;
		counter < planeCount/gridDim.x;
		i += gridStride)
	{
		sum = 0;
		
		//float sum =0;
		//calculate the first target index
 		

 		/*
		target = (i%dim1) * s1;
 		target += ((i/ dim1) % dim2) * s2;
 		target += ((i/area)) * splane;
 		*/
 		//printf("Test: tid= %d  target= %d target2= %d \n\n", i, target, target + (dim2/2 * s2));
 		
 		for(int iter=0; iter < noEls; iter++)
 		{
 			sum += in[i + iter*quarter];

 		}
 		//__syncthreads();
 		/*
 		sum = in[gridStride*counter + target] + in[gridStride*counter + target + quarter] +
 				in[gridStride*counter + target + 2*quarter] + in[gridStride*counter+target+ 3*quarter];
 		*/		
 		//sum += in[i] + in[i+blockDim.x];
 		sum = blockReduceSum(sum);
		if(threadIdx.x == 0)
			out[counter*gridDim.x + blockIdx.x] = sum;
 		
 		counter++;
 		//sum += in[i] + in[i + blockDim.x];
	}
	
}

void run_test(int noEls, int noOfBlocks, int r1, int r2, int r3, int rplane, int dimen1, int dimen2, int dimen3, int dimen4, int dimen5)
{

	
	const int dim_len = 5;

	//dimension sizes
	int dims[dim_len] = {dimen1,dimen2,dimen3,dimen4,dimen5};
	//dimensions to reduce
	//int rdims[2] = {r1,r2}; //x and y
	int rdims[3] = {r1,r2,r3};

	int strides[dim_len];

	strides[0] = 1;

	//total number of elements
	int N = dims[0];




	for(int i=1; i<dim_len; i++){
		strides[i] = dims[i-1] * strides[i-1]; 
		//update N
		N *= dims[i];
	}
	//N = dimen1 * dimen2 * dimen3;
	printf("Number of elements: %d\n\n", N);
	printArr(strides);

	//Allocate memory for in and out in host and fill in
	double *in, *out, *d_in, *d_out;
	//int *d_strides, *d_rdims, *d_dims;

	in = (double*)malloc(N*sizeof(double));

	
	int planeCount = int(N)/(dims[rdims[0]]*dims[rdims[1]]*dims[rdims[2]]);//dims[rplane]*dims[4];//8192;//131072;

	out = (double*)malloc(planeCount*sizeof(double)); 
	srand(time(NULL));
	for(int i=0; i<N;i++)
	{
		if((i/128) %16384 == 16300)
		{
			in[i] =  double(i)/1000;//(float)rand() / (float)RAND_MAX;////
		}
		else
		{
			in[i] = double(i)/1000;//(float)rand() / (float)RAND_MAX;
		}
	}
	/*
	for(int i=0;i<N;i++)
	{
		printf("%.1f ", in[i]);
	}
	*/
	printf("\n\n");
	//Allocate memory for in and out on device
	hipMalloc(&d_in, N*sizeof(double));
	hipMalloc(&d_out, planeCount*sizeof(double));
	

	//Event variables
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	
	//Transfer host data to device

	gpuErrchk(hipMemcpy(d_in, in, N*sizeof(double), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_out, out, planeCount*sizeof(double), hipMemcpyHostToDevice));


	int s1 = strides[rdims[0]];
	int s2 = strides[rdims[1]];
	int s3 = strides[rdims[2]];
	
	int splane = strides[rplane];
	int dim1 = dims[rdims[0]];
	int dim2 = dims[rdims[1]];
	int dim3 = dims[rdims[2]];
	
	//int noElems = noEls;
	//Record kernel
	int noMeasures = 100; //number of measurements to take
	hipEventRecord(start);
	for(int mesIter=0; mesIter<noMeasures;mesIter++)
	{
		reduction<<<noOfBlocks,((dim1*dim2*dim3)/noEls)>>>(d_in,d_out, N, s1,s2,s3, splane, dim1,dim2,dim3, planeCount, noEls);
	}
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms,start,stop);

	gpuErrchk(hipMemcpy(out, d_out, planeCount*sizeof(double), hipMemcpyDeviceToHost));
	ms = ms/noMeasures;
	double total = (N)*8;

	double ebw = total/(ms*1e6);

	printf("EBW: %f\n", ebw);
	//Check errors
	printf("Strides: %d  ", s1);
	printf("Plane Stride: %d\n ", splane);
	printf("Plane Count: %d\n", planeCount);

	for(int i=0;i<4;i++)
	{
		
		printf("%d : %.3f  ", i, out[i]);
	}
	double sizeOut = sizeof(out);
	double sizeD = sizeof(double);
	int lengthOut = sizeOut/sizeD;
	printf("Length: %d\n", lengthOut);
	printf("%.3f", out[16300/*131071*/]);
	printf("\n");
	hipFree(d_in);
	hipFree(d_out);
}

int main(int argc, char *argv[])
{
	{
		int noEls = 8;
		int noOfBlocks = 512;
		int r1 = 0;
		//int r2 = 1;
		int r2 = 1;
		int r3 = 2;
		int rplane = 3;
		int dim1 = 8;
		int dim2 = 16;
		int dim3 = 16;
		int dim4 = 32;
		int dim5 = 32;
		if(argc > 1)
		{
			noEls = atoi(argv[1]);
			noOfBlocks = atoi(argv[2]);
			r1 = atoi(argv[3]);
			r2 = atoi(argv[4]);
			r3 = atoi(argv[5]);
			rplane = atoi(argv[6]);
			dim1 = atoi(argv[7]);
			dim2 = atoi(argv[8]);
			dim3 = atoi(argv[9]);
			dim4 = atoi(argv[10]);
			dim5 = atoi(argv[11]);
		}
		run_test(noEls, noOfBlocks, r1, r2, r3, rplane, dim1, dim2, dim3, dim4, dim5);
	}
	return 0;
}

void printArr(int *arr)
{
	int i;
	printf("Stride values in order: ");
	for(i=0;i<=sizeof(arr)/sizeof(int);i++)
	{

		printf("%d ", arr[i]);
	}
	printf("\n\n");
}

