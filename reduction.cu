

#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <limits.h>
#include <stdlib.h>

void run_test();
void printArr(int *arr);

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        printf("GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__inline__ __device__
float warpReduceSum(float val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2) 
    val += __shfl_down(val, offset);
  return val;
}

__inline__ __device__
float blockReduceSum(float val) {

  static __shared__ float shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSum(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

  if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

  return val;
}

__global__ void reduction(float *in, float *out, int N, int s1, int s2, int splane, int dim1, int dim2, int planeCount)
{
	float sum =0;
	int cur_plane;

	int start = blockIdx.x * 8 * blockDim.x + threadIdx.x;
	int gridStride = blockDim.x * 8 * gridDim.x;

	//relative index and coordinates calculation
	int area = dim1 * dim2;
	
	int target = (start%dim1) * s1;
 	target += ((start / dim1) % dim2) * s2;
 	target += ((start/area)%planeCount) * splane;
	/*int tempDiv = area;
	/*
	for(int dimIter=0; dimIter<noDims; dimIter++)
	{
		if(dimIter != noDims-1)
		{
			dCoord = start / tempDiv % dimSizes[dimIter];
		}
	}
	*/
	int counter = 0;
	int quarter = dim2 / 8;
	quarter = quarter *s2;
	for(int i = start;
		i < N;
		i += gridStride)
	{
		sum = 0;
		
		//float sum =0;
		//calculate the first target index
 		

 		//determine which plane the thread is reducing
 		//cur_plane = (int)(i/area);

 		//printf("Test: tid= %d  target= %d target2= %d \n\n", i, target, target + (dim2/2 * s2));
 		
 		for(int iter=0; iter < 8; iter++)
 		{
 			sum += in[gridStride*counter + target + iter*quarter];

 		}
 		//__syncthreads();
 		/*
 		sum = in[gridStride*counter + target] + in[gridStride*counter + target + quarter] +
 				in[gridStride*counter + target + 2*quarter] + in[gridStride*counter+target+ 3*quarter];
 		*/		
 		//sum += in[i] + in[i+blockDim.x];
 		sum = blockReduceSum(sum);
		if(threadIdx.x == 0)
			out[counter*gridDim.x + blockIdx.x] = sum;
 		
 		counter++;
 		//sum += in[i] + in[i + blockDim.x];
	}
	
}



int main(void)
{
	{
		run_test();
	}
	return 0;
}

void printArr(int *arr)
{
	int i;
	printf("Stride values in order: ");
	for(i=0;i<=sizeof(arr)/sizeof(int);i++)
	{

		printf("%d ", arr[i]);
	}
	printf("\n\n");
}
void run_test()
{

	printf("%.3f\n", float(20/1000) );
	const int dim_len = 4;

	//dimension sizes
	int dims[dim_len] = {32,32,1024,256};
	//dimensions to reduce
	int rdims[2] = {0,1}; //x and y


	int strides[dim_len];

	strides[0] = 1;

	//total number of elements
	int N = dims[0];




	for(int i=1; i<dim_len; i++){
		strides[i] = dims[i-1] * strides[i-1]; 
		//update N
		N *= dims[i];
	}
	printf("Number of elements: %d\n\n", N);
	printArr(strides);

	//Allocate memory for in and out in host and fill in
	float *in, *out, *d_in, *d_out;
	//int *d_strides, *d_rdims, *d_dims;

	in = (float*)malloc(N*sizeof(float));

	
	int planeCount = 1024*256;//8192;//131072;

	out = (float*)malloc(planeCount*sizeof(float)); 
	srand(time(NULL));
	for(int i=0; i<N;i++)
	{
		in[i] = (float)rand() / (float)RAND_MAX; //float(i)/float(1000);//
	}
	/*
	for(int i=0;i<N;i++)
	{
		printf("%.1f ", in[i]);
	}
	*/
	printf("\n\n");
	//Allocate memory for in and out on device
	hipMalloc(&d_in, N*sizeof(float));
	hipMalloc(&d_out, planeCount*sizeof(float));
	

	//Event variables
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	
	//Transfer host data to device

	gpuErrchk(hipMemcpy(d_in, in, N*sizeof(float), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_out, out, planeCount*sizeof(float), hipMemcpyHostToDevice));


	int s1 = strides[rdims[0]];
	int s2 = strides[rdims[1]];
	int splane = strides[2];
	int dim1 = dims[rdims[0]];
	int dim2 = dims[rdims[1]];
	int noEls = 8;
	//Record kernel
	int noMeasures = 10; //number of measurements to take
	hipEventRecord(start);
	for(int mesIter=0; mesIter<noMeasures;mesIter++)
	{
		reduction<<<4096,128>>>(d_in,d_out, N, s1, s2, splane, dim1, dim2, planeCount);
	}
	
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float ms = 0;
	hipEventElapsedTime(&ms,start,stop);

	gpuErrchk(hipMemcpy(out, d_out, planeCount*sizeof(float), hipMemcpyDeviceToHost));
	ms = ms/noMeasures;
	double total = (N+planeCount)*4;

	double ebw = total/(ms*1e6);

	printf("EBW: %f\n", ebw);
	//Check errors
	printf("Strides: %d %d ", s1,s2);
	printf("Plane Stride: %d\n ", splane);
	printf("Plane Count: %d\n", planeCount);

	for(int i=0;i<4;i++)
	{
		
		printf("%.3f %d ", out[i], i);
	}
	printf("%.3f", out[63/*131071*/]);
	printf("\n");
	hipFree(d_in);
	hipFree(d_out);
}